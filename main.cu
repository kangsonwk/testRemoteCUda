
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloCUDA()
{
    printf("GPU：Hello, CUDA!\n");
}

int main()
{
    helloCUDA<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}